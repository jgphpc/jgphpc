// --- CSCS (Swiss National Supercomputing Center) ---
// samples/1_Utilities/deviceQuery/deviceQuery.cpp

#include <stdio.h>
#include <hip/hip_runtime_api.h>

extern "C"
void set_gpu(int dev)
{
  hipSetDevice(dev);
}

extern "C"
void get_gpu_info(char *gpu_string, int dev)
{
  struct hipDeviceProp_t dprop;
  hipGetDeviceProperties(&dprop, dev);
  strcpy(gpu_string,dprop.name);
}

extern "C"
void get_more_gpu_info(int dev)
{
  int driverVersion = 0, runtimeVersion = 0;
  struct hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);

  printf("Device %d: \"%s\"\n", dev, deviceProp.name);
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("  CUDA Driver Version / Runtime Version     %d.%d / %d.%d\n", 
    driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, 
    (runtimeVersion%100)/10);

  printf("  CUDA Capability Major/Minor version number:    %d.%d\n", 
    deviceProp.major, deviceProp.minor);

  printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
  deviceProp.multiProcessorCount,
  _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
  _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) 
                            * deviceProp.multiProcessorCount);

  printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", 
    deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
  printf("  Theoretical peak performance per GPU:          %.0f Gflop/s\n",
    deviceProp.clockRate *1e-6f 
    *_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor)
    *deviceProp.multiProcessorCount);

//  printf("  Maximum number of threads per multiprocessor:  %d\n", 
//    deviceProp.maxThreadsPerMultiProcessor);

  printf("  Peak number of threads:                        %d threads\n", 
    deviceProp.multiProcessorCount 
    * deviceProp.maxThreadsPerMultiProcessor );

  printf("  Maximum number of threads per block:           %d\n", 
    deviceProp.maxThreadsPerBlock);

  // --- deviceQuery.cpp ---
  // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DEVICE.html
  printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
  printf("  Warp size: %d\n", deviceProp.warpSize);    
  printf("  Maximum number of threads per multiprocessor:  %d\n",
          deviceProp.maxThreadsPerMultiProcessor);
  printf("  Maximum number of threads per block:           %d\n",
          deviceProp.maxThreadsPerBlock);
  printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
          deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
          deviceProp.maxThreadsDim[2]);
  printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
          deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
          deviceProp.maxGridSize[2]);
  printf("  Device has ECC support:                        %s\n",
          deviceProp.ECCEnabled ? "Enabled" : "Disabled");
  printf("  multiProcessorCount:                           %d\n",
          deviceProp.multiProcessorCount);

  // summary table
  printf("%10s| %7s | %7s | %7s | %7s |\n", " ", "thread", "warp", "sm", "device");
  printf("threads   | %7d | %7d | %7d | %7d |\n", 
          1, deviceProp.warpSize, deviceProp.maxThreadsPerMultiProcessor, 
          deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor);
  printf("warps     | %7s | %7d | %7d | %7d |\n", "x", 1,
          deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize,
          (deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor) / deviceProp.warpSize);
  printf("sms       | %7s | %7s | %7d | %7d |\n", "x", "x", 1, deviceProp.multiProcessorCount);
  printf("%-.10s| %7s | %7s | %7s | %7d |\n", deviceProp.name, "x", "x", "x", 1);

/*
          |  thread |    warp |      sm |  device |
threads   |       1 |      32 |    2048 |  114688 |
warps     |       x |       1 |      64 |    3584 |
sms       |       x |       x |       1 |      56 |
Tesla P100|       x |       x |       x |       1 |
*/

}

